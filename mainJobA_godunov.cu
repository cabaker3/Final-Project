#include "hip/hip_runtime.h"
#include <iostream>
#include "omp.h"
#include "helperJobB_kernel.cu"
#include <math.h>
#include <utility>
#include <hip/hip_runtime.h>

using namespace std;

//run on the host using OpenMP

void mainJobA_godunov(int L, float g, float dx, float dt, float IM){
  //constant
  int i = 1;
  const int row1 = 101;
  const int row2 = 3;
  const int col1 = 1;
  const int col2 = 101;
  const int col3 = 81;
  
  //create 2D array Qi
  float **Qi = new float*[row2];
  for(int i = 0; i < row2; ++i) {
    Qi[i] = new float[col2];
  }
  
  //Lax Initial Conditions
  for(int x = 0; x <= 1; x+=dx){
    if(x <= 0.5){
      Qi[1][i] = 0.445;
      Qi[2][i] = 0.311;
      Qi[3][i] = 8.928;
    }else{
      Qi[1][i] = 0.5;  
      Qi[2][i] = 0;
      Qi[3][i] = 1.4275;
    }
    i += 1; //change to for loop
  }
  
  //create array Qold = Qi
  float **Qold = new float*[row2];
  float **Qnew = new float*[row2];
  for(int i = 0; i < row2; ++i) {
    Qold[i] = new float[col2];
    Qnew[i] = new float[col2];
  }
  memcpy(Qold, Qi, sizeof(Qold));
  //create array Qnew = Qold
  memcpy(Qnew, Qold, sizeof(Qnew));
  
  //Initial Flow Properties
  
  
  //create array rhoi, ui, eti, pi, a, E, eigen
  float **rhoi = new float*[row1];
  float **ui = new float*[row1];
  float **eti = new float*[row1];
  float **pi = new float*[row1];
  float **a = new float*[row1];
  float **alpha = new float*[row1];
  for(int i = 0; i < row1; ++i) {
    rhoi[i] = new float[col1];
    ui[i] = new float[col1];
    eti[i] = new float[col1];
    pi[i] = new float[col1];
    a[i] = new float[col1];
    alpha[i] = new float[col1];
  }
  
  float **E = new float*[row2];
  float **eigen = new float*[row2];
  for(int i = 0; i < row2; ++i) {
    E[i] = new float[col2];
    eigen[i] = new float[col2];
  }
  
  #pragma omp for
  for(int i = 1; i < IM+1; i++){
    //Density
    rhoi[i][1] = Qi[1][i];
    
    //Velocity
    ui[i] = Qi[2][i] / rhoi[i][1];
    
    //Total Energy
    eti[i][1] = Qi[3][i] / rhoi[i][1];
    
    //Pressure, from the equation of state
    pi[i][1] = (g-1) * (rhoi[i][1]) * eti[i][1] - 0.5 * rhoi[i][1] * pow(ui[i][1],2));
      
    //Speed of Sound
    a[i][1] = sqrt(g*pi[i][1]/rhoi[i][1]);
    
    //Intial E Matrix
    E[1][i] = rhoi[i][1]*ui[i][1];
    E[2][i] = rhoi[i][]*pow(ui[i][1],2) + pi[i][1];
    E[3][i] = eti[i][]*rhoi[i][1]*ui[i][1]+pi[i][1]*ui[i][1]};
    
    //Eigenvalues
    eigen[1][i] = ui[i][1];
    eigen[2][i] = ui[i][1] + a[i][1]; 
    eigen[3][i] = ui[i][1]-a[i][1];
  }
  
  #pragma omp for
  for(int i = 0; i < row1; ++i) {
    alpha[i][1] = max(abs(eigen));
  }
  
  delete[] rhoi;
  delete[] ui;
  delete[] eti;
  delete[] pi;
  delete[] a;
  delete[] E;
  delete[] eigen;
  
  int k = 1;
  
  //initialize rho,u,et,p,a,E,eigen, F,Qn1
  auto rho = new float [row1][col1];
  auto u = new float [row1][col1];
  auto et = new float [row1][col1];
  auto p = new float [row1][col1];
  auto a = new float [row1][col1];
  auto E = new float [row2][col2];
  auto eigen = new float [row2][col2];
  auto F = new float [row2][100];
  auto Qn1 = new float [row2][col2];
  auto rhom = new float [row1][col3];
  auto um = new float [row1][col3];
  auto etm = new float [row1][col3];
  auto pm = new float [row1][col3];
  
  #pragma omp for
  for(int t = 0; t <= 0.16; t+=dt){
    //call helperJobB
    memcpy(F,helperJobB_godunov(alpha,E,Qold,Qnew,F,IM),sizeof(F)); //Flux
    
    #pragma omp for
    for(int j = 1; j <= 3; j++){
      for(int i = 2; i <= IM; i++){
        Qn1[j][i] = Qold[j][i] - (dt/dx) * (F[j][i] - F[j][i-1]); 
      }
    }
    
    Qn1[][1] = Qi[][1];
    Qn1[][101] = Qi[][101];
    Qnew = Qn1;
    Qold = Qnew;
    
    #pragma omp for
    for(int i = 1; i < IM+1; i++){
    //Density
    rho[i][] = Qnew[1][i];
    
    //Velocity
    u[i] = Qnew[2][i] / rho[i][];
    
    //Total Energy
    et[i][] = Qnew[3][i] / rho[i][];
    
    //Pressure, from the equation of state
    p[i][] = (g-1) * (rho[i][] * et[i][] - 0.5 * rho[i][] * pow(u[i][],2));
      
    //Speed of Sound
    a[i][] = sqrt(g*p[i][]/rho[i][]);
    
    //Intial E Matrix
    E[][i] = {rho[i][]*u[i][], rho[i][]*pow(ui[i][],2) + p[i][], et[i][]*rho[i][]*u[i][]+p[i][]*u[i][]};
    
    //Eigenvalues
    eigen[][i] = {u[i][], u[i][] + a[i][], u[i][]-a[i][]};
  }
    //Alpha
    alpha = max(abs(eigen));
    
    um[][k] =u[][1];
    rhom[][k] = rho[][1];
    pm[][k] = p[][1];
    etm[][k] = et[][1];
    
    k += 1;
  }
  
  delete[] rho;
  delete[] u;
  delete[] et;
  delete[] p;
  delete[] a;
  delete[] E;
  delete[] eigen;
  delete[] alpha;
  delete[] F;
  delete[] Qn1;
  delete[] Qi;
  delete[] Qnew;
  delete[] Qold;
  
  //cout
  cout << rhom << "\n" << um << "\n" << etm << "\n" << pm << "\n";
  
  delete[] rhom;
  delete[] um;
  delete[] etm;
  delete[] pm;
  //return ;
}
