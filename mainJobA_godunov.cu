#include "hip/hip_runtime.h"
#include <iostream>
#include "omp.h"
#include "helperJobB_kernel.cu"
#include <math.h>
#include <utility>
#include <hip/hip_runtime.h>

using namespace std;

//run on the host using OpenMP

void mainJobA_godunov(int L, float g, float dx, float dt, float IM){
  //constant
  int i = 1;
  const int row1 = 101;
  const int row2 = 3;
  const int col1 = 1;
  const int col2 = 101;
  const int col3 = 81;
  dim3 dimBlock(3*100, 3*100);
  dim3 dimGrid(1, 1);
  
  //create 2D array Qi
  float **Qi = new float*[row2];
  for(int i = 0; i < row2; ++i) {
    Qi[i] = new float[col2];
  }
  
  //Lax Initial Conditions
  for(int x = 0; x <= 1; x+=dx){
    if(x <= 0.5){
      Qi[1][i] = 0.445;
      Qi[2][i] = 0.311;
      Qi[3][i] = 8.928;
    }else{
      Qi[1][i] = 0.5;  
      Qi[2][i] = 0;
      Qi[3][i] = 1.4275;
    }
    i += 1; //change to for loop
  }
  
  //create array Qold = Qi
  float **Qold = new float*[row2];
  float **Qnew = new float*[row2];
  for(int i = 0; i < row2; ++i) {
    Qold[i] = new float[col2];
    Qnew[i] = new float[col2];
  }
  memcpy(Qold, Qi, sizeof(Qold));
  //create array Qnew = Qold
  memcpy(Qnew, Qold, sizeof(Qnew));
  
  //Initial Flow Properties
  
  
  //create array rhoi, ui, eti, pi, a, E, eigen
  float **rhoi = new float*[row1];
  float **ui = new float*[row1];
  float **eti = new float*[row1];
  float **pi = new float*[row1];
  float **a = new float*[row1];
  float **alpha = new float*[row1];
  for(int i = 0; i < row1; ++i) {
    rhoi[i] = new float[col1];
    ui[i] = new float[col1];
    eti[i] = new float[col1];
    pi[i] = new float[col1];
    a[i] = new float[col1];
    alpha[i] = new float[col1];
  }
  
  float **E = new float*[row2];
  float **eigen = new float*[row2];
  for(int i = 0; i < row2; ++i) {
    E[i] = new float[col2];
    eigen[i] = new float[col2];
  }
  
  #pragma omp for
  for(int i = 1; i < IM+1; i++){
    //Density
    rhoi[i][1] = Qi[1][i];
    
    //Velocity
    ui[i][1] = Qi[2][i] / rhoi[i][1];
    
    //Total Energy
    eti[i][1] = Qi[3][i] / rhoi[i][1];
    
    //Pressure, from the equation of state
    pi[i][1] = (g-1) * (rhoi[i][1]) * eti[i][1] - 0.5 * rhoi[i][1] * pow(ui[i][1],2));
      
    //Speed of Sound
    a[i][1] = sqrt(g*pi[i][1]/rhoi[i][1]);
    
    //Intial E Matrix
    E[1][i] = rhoi[i][1]*ui[i][1];
    E[2][i] = rhoi[i][1]*pow(ui[i][1],2) + pi[i][1];
    E[3][i] = eti[i][1]*rhoi[i][1]*ui[i][1]+pi[i][1]*ui[i][1]};
    
    //Eigenvalues
    eigen[1][i] = ui[i][1];
    eigen[2][i] = ui[i][1] + a[i][1]; 
    eigen[3][i] = ui[i][1]-a[i][1];
  }
  
  #pragma omp for
  for(int i = 0; i < row1; ++i) {
    alpha[i][1] = max(abs(eigen));
  }
  
  delete[] rhoi;
  delete[] ui;
  delete[] eti;
  delete[] pi;
  delete[] a;
  delete[] E;
  delete[] eigen;
  
  int k = 1;
  
  //initialize rho,u,et,p,a,E,eigen, F,Qn1
  float **rho = new float*[row1];
  float **u = new float*[row1];
  for(int i = 0; i < row1; ++i) {
    rho[i] = new float[col1];
    u[i] = new float[col1];
    et[i] = new float[col1];
    p[i] = new float[col1];
    a[i] = new float[col1];
  }
 
  float **E = new float*[row2];
  float **eigen = new float*[row2];
  float **Qn1 = new float*[row2];
  for(int i = 0; i < row2; ++i) {
    E[i] = new float[col2];
    eigen[i] = new float[col2];
    Qn1[i] = new float[col2];
  }

  float **rhom = new float*[row1];
  float **um = new float*[row1];
  float **etm = new float*[row1];
  float **pm = new float*[row1];
  for(int i = 0; i < row1; ++i) {
    rhom[i] = new float[col3];
    um[i] = new float[col3];
    etm[i] = new float[col3];
    pm[i] = new float[col3];
  }

  float **F = new float*[row2];
  for(int i = 0; i < row2; ++i) {
    F[i] = new float[100];
  }
  
  #pragma omp for
  for(int t = 0; t <= 0.16; t+=dt){
    //call helperJobB
    helperJobB_godunov<<<dimGrid, dimBlock>>>(alpha,E,Qold,Qnew,F,IM); //Flux
    
    #pragma omp barrier
    for(int j = 1; j <= 3; j++){
      for(int i = 2; i <= IM; i++){
        Qn1[j][i] = Qold[j][i] - (dt/dx) * (F[j][i] - F[j][i-1]); 
      }
    }
    
    for(int x = 0; x <= 3; x++){
    Qn1[x][1] = Qi[x][1];
    Qn1[x][101] = Qi[x][101];
    }
    
    memcpy(Qnew, Qn1, sizeof(Qnew));
    memcpy(Qold, Qnew, sizeof(Qold));
    
    #pragma omp for
    for(int i = 1; i < IM+1; i++){
    //Density
    rho[i][1] = Qnew[1][i];
    
    //Velocity
    u[i] = Qnew[2][i] / rho[i][1];
    
    //Total Energy
    et[i][1] = Qnew[3][i] / rho[i][1];
    
    //Pressure, from the equation of state
    p[i][1] = (g-1) * (rho[i][1] * et[i][1] - 0.5 * rho[i][1] * pow(u[i][1],2));
      
    //Speed of Sound
    a[i][1] = sqrt(g*p[i][1]/rho[i][1]);
    
    //Intial E Matrix
    E[1][i] = rho[i][1]*u[i][1];
    E[2][i] = rho[i][1]*pow(ui[i][1],2) + p[i][1];
    E[3][i] = et[i][1]*rho[i][1]*u[i][1]+p[i][1]*u[i][1];
    
    //Eigenvalues
    eigen[1][i] = u[i][1];
    eigen[2][i] = u[i][1] + a[i][1]; 
    eigen[3][i] = u[i][1]-a[i][1];
  }
    //Alpha
    alpha = max(abs(eigen));
    
    for(int x = 0; x <= 101; x++){
      um[x][k] =u[x][1];
      rhom[x][k] = rho[x][1];
      pm[x][k] = p[x][1];
      etm[x][k] = et[x][1];
    }
    
    k += 1;
  }
  
  delete[] rho;
  delete[] u;
  delete[] et;
  delete[] p;
  delete[] a;
  delete[] E;
  delete[] eigen;
  delete[] alpha;
  delete[] F;
  delete[] Qn1;
  delete[] Qi;
  delete[] Qnew;
  delete[] Qold;
  
  // display properties matrices 
  cout << rhom << "\n" << um << "\n" << etm << "\n" << pm << "\n";
  
  delete[] rhom;
  delete[] um;
  delete[] etm;
  delete[] pm;
}
