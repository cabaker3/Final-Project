#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__ void helperJobB(float *alpha, float *E, float *Qold, float *Qnew, float *F, float IM){
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  
  for(int j = 1; j <= 3; j++){
    for(int i = 1; i <= IM; i++)
      F[j][i] = 0.5 * (E[j][i] + E[j][i+1]) - 0.5 * abs(alpha[i][1]) * (Qold[j][i+1] - Qnew[j][i]);
  }
  
}
